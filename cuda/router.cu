#include <router.h>

#define MEASURE_LATENCY
#define MEASURE_BANDWIDTH
#define MEASURE_PROCESSING_MICROBENCHMARK

#define DEFAULT_BLOCK_SIZE 32

bool do_run = true;


/**
 * Start a loop:
 *	1) Gather packets
 *  2) Copy packets to GPU and process
 *	3) Copy results back and print performance stats
 *
 * We do this with pipelining: while the GPU is processing one buffer of packets,
 * we're copying over the next batch so that it can begin processing them as soon
 * as it finishes processing the first batch.
 */
int run(int argc, char **argv, int block_size, int sockfd)
{
	printf("Running CPU/GPU code\n\n");

	unsigned int buf_size = sizeof(packet)*get_batch_size();
	unsigned int results_size = sizeof(int)*get_batch_size();

	// Allocate host memory for two batches of up to batch_size packets
	// We will alternate between filling and processing these two buffers
	// (at any given time one of the buffers will either be being filled
	// or being processed)
	packet* h_p1 = (packet *)malloc(buf_size);
	check_malloc(h_p1, "h_p1", __LINE__);
	packet* h_p2 = (packet *)malloc(buf_size);
	check_malloc(h_p2, "h_p2", __LINE__);

	// Allocate host memory for 2 arrays of results
	int *h_results1 = (int*)malloc(results_size);
	check_malloc(h_results1, "h_results1", __LINE__);
	int *h_results2 = (int*)malloc(results_size);
	check_malloc(h_results2, "h_results2", __LINE__);

	// Allocate device memory for up to batch_size packets
	// TODO: wait and allocate only the amount needed after we receive?
	packet *d_p1;
	check_error(hipMalloc((void **) &d_p1, buf_size), "hipMalloc d_p1", __LINE__);
	packet *d_p2;
	check_error(hipMalloc((void **) &d_p2, buf_size), "hipMalloc d_p2", __LINE__);
	// Allocate device memory for results
	int *d_results1;
	check_error(hipMalloc((void **) &d_results1, results_size), "hipMalloc d_results1", __LINE__);
	int *d_results2;
	check_error(hipMalloc((void **) &d_results2, results_size), "hipMalloc d_results2", __LINE__);


	// Setup execution parameters
	int threads_in_block = block_size;
	int blocks_in_grid = get_batch_size() / threads_in_block;  // FIXME: optimize if we don't have a full batch
	if (get_batch_size() % threads_in_block != 0) {
		blocks_in_grid++;  // need an extra block for the extra threads
	}


	// Run any processing-specific setup code needed
	// (e.g., this might copy the FIB to GPU for LPM)
	setup();
   

#ifdef MEASURE_PROCESSING_MICROBENCHMARK
	// Allocate CUDA events that we'll use for timing
	hipEvent_t start;
	check_error(hipEventCreate(&start), "Create start event", __LINE__);

	hipEvent_t stop;
	check_error(hipEventCreate(&stop), "Create stop event", __LINE__);
#endif /* MEASURE_PROCESSING_MICROBENCHMARK */

#ifdef MEASURE_BANDWIDTH
	long packets_processed = 0;
	struct timeval bw_start, bw_stop;
	gettimeofday(&bw_start, NULL);
#endif /* MEASURE_BANDWIDTH */

#ifdef MEASURE_LATENCY
	double max_latency, min_latency;
	struct timeval lat_start_oldest1, lat_start_oldest2, lat_start_newest1, lat_start_newest2, lat_stop;
	struct timeval *lat_start_oldest_current = &lat_start_oldest1;
	struct timeval *lat_start_oldest_next = &lat_start_oldest2;
	struct timeval *lat_start_newest_current = &lat_start_newest1;
	struct timeval *lat_start_newest_next = &lat_start_newest2;
#endif /* MEASURE_LATENCY */

	bool data_ready = false;
	bool results_ready = false;
	int num_packets;
	packet *h_p_current = h_p1;
	packet *h_p_next = h_p2;
	packet *d_p_current = d_p1;
	packet *d_p_next = d_p2;
	int *h_results_current = h_results1;
	int *h_results_previous = h_results2;
	int *d_results_current = d_results1;
	int *d_results_previous = d_results2;
	
	/* The main loop:
		1) Execute the CUDA kernel
		2) While it's executing, copy the results from the last execution back to the host
		3) While it's executing, copy the packets for the next execution to the GPU */
	
	while(do_run) {
		
		/*************************************************************
		 *				1) EXECUTE THE CUDA KERNEL				 *
		 *************************************************************/
		if (data_ready) { // First execution of loop: data_ready = false

#ifdef MEASURE_PROCESSING_MICROBENCHMARK
			// Record the start event
			check_error(hipEventRecord(start, NULL), "Record start event", __LINE__);
#endif /* MEASURE_PROCESSING_MICROBENCHMARK */

			// Execute the kernel
			printf("vvvvv   Begin processing %d packets   vvvvv\n\n", num_packets);
			process_packets<<< blocks_in_grid, threads_in_block >>>(d_p_current, d_results_current, num_packets, block_size);

#ifdef MEASURE_PROCESSING_MICROBENCHMARK
			// Record the stop event
			check_error(hipEventRecord(stop, NULL), "Record stop event", __LINE__);
#endif /*MEASURE_PROCESSING_MICROBENCHMARK*/


#ifdef MEASURE_BANDWIDTH
			packets_processed += num_packets;
#endif /* MEASURE_BANDWIDTH */

		}


		
		/*************************************************************
		 *			2) COPY BACK RESULTS FROM LAST BATCH		   *
		 *************************************************************/
		if (results_ready) { // First and second executions of loop: results_ready = false

			// TODO: double-check that stuff is really executing when I think it is.
			// I think that calling hipEventRecord(stop) right before this will record
			// when the kernel stops executing, but won't block until this actually happens.
			// The hipEventSynchronize call below does block until the kernel stops.
			// So, I think anything we do here will execute on the CPU while the GPU executes
			// the kernel call we made above.

			// Copy the last set of results back from the GPU
			check_error(hipMemcpy(h_results_previous, d_results_previous, results_size, hipMemcpyDeviceToHost), "hipMemcpy (h_results, d_results)", __LINE__);

#ifdef MEASURE_LATENCY
			gettimeofday(&lat_stop, NULL);
			max_latency = (lat_stop.tv_sec - lat_start_oldest_current->tv_sec) * 1000000.0 + (lat_stop.tv_usec - lat_start_oldest_current->tv_usec);
			min_latency = (lat_stop.tv_sec - lat_start_newest_current->tv_sec) * 1000000.0 + (lat_stop.tv_usec - lat_start_newest_current->tv_usec);
			printf("Latencies from last batch: Max: %f msec   Min: %f msec\n", max_latency, min_latency);
#endif /* MEASURE_LATENCY */
		
			// Print results
			printf("Results from last batch:\n");
			int i;
			for (i = 0; i < get_batch_size(); i++) {
				printf("%d, ", h_results_previous[i]);
			}
			printf("\n\n");
		}

		
		
		
		/*************************************************************
		 *				  3) COPY NEXT BATCH TO GPU				*
		 *************************************************************/
		// Get next batch of packets and copy them to the GPU
		// FIXME: We're forcing the results from the current execution to wait
		// until we get the next batch of packets. Is this OK?
#ifdef MEASURE_LATENCY
		// Approx time we received the first packet of the batch
		// (not perfect if the first packet doesn't arrive immediately)
		gettimeofday(lat_start_oldest_next, NULL);
#endif /* MEASURE_LATENCY */
		num_packets = 0;
		while (num_packets == 0 && do_run) {
			num_packets = get_packets(sockfd, h_p_next);
		}
#ifdef MEASURE_LATENCY
		gettimeofday(lat_start_newest_next, NULL);
#endif /* MEASURE_LATENCY */
		check_error(hipMemcpy(d_p_next, h_p_next, buf_size, hipMemcpyHostToDevice), "hipMemcpy (d_p_next, h_p_next)", __LINE__);



		if (data_ready) {

#ifdef MEASURE_PROCESSING_MICROBENCHMARK
			// Wait for the stop event to complete (which waits for the kernel to finish)
			check_error(hipEventSynchronize(stop), "Failed to synchronize stop event", __LINE__);
			
			float msecTotal = 0.0f;
			check_error(hipEventElapsedTime(&msecTotal, start, stop), "Getting time elapsed b/w events", __LINE__);

			// Compute and print the performance
			printf(
				"Performance= Time= %.3f msec, WorkgroupSize= %u threads/block\n",
				msecTotal,
				threads_in_block);
#else
			// Wait for kernel execution to complete
			check_error(hipDeviceSynchronize(), "hipDeviceSynchronize", __LINE__);
#endif /* MEASURE_PROCESSING_MICROBENCHMARK */


			printf("^^^^^   Done processing batch   ^^^^^\n\n\n");

			results_ready = true;
		}
		data_ready = true;




		// Get ready for the next loop iteration
		SWAP(h_p_current, h_p_next, packet*);
		SWAP(d_p_current, d_p_next, packet*);
		SWAP(h_results_current, h_results_previous, int*);
		SWAP(d_results_current, d_results_previous, int*);
#ifdef MEASURE_LATENCY
		SWAP(lat_start_oldest_current, lat_start_oldest_next, struct timeval*);
		SWAP(lat_start_newest_current, lat_start_newest_next, struct timeval*);
#endif /* MEASURE_LATENCY */

	}


#ifdef MEASURE_BANDWIDTH
	// Calculate how many packets we processed per second
	gettimeofday(&bw_stop, NULL);
	double total_time = (bw_stop.tv_sec - bw_start.tv_sec) + (bw_stop.tv_usec - bw_start.tv_usec) / 1000000.0;
	double pkts_per_sec = double(packets_processed) / total_time;	

	printf("Bandwidth: (%ld/%f) = %f packets per second\n", packets_processed, total_time, pkts_per_sec);
#endif /* MEASURE_BANDWIDTH */




	// Clean up memory
	free(h_p1);
	free(h_p2);
	free(h_results1);
	free(h_results2);
	hipFree(d_p1);
	hipFree(d_p2);
	hipFree(d_results1);
	hipFree(d_results2);

	hipDeviceReset();

	return EXIT_SUCCESS;
}

void test(int sockfd) 
{
	printf("Batch Size: %d\n", get_batch_size());
	
	// Initialize a buffer for storing up to batch_size packets
	packet* p = (packet *)malloc(sizeof(packet)*get_batch_size());
	
	while(1) {
	  int num_packets = get_packets(sockfd, p);
	  printf("i = %d\n", num_packets);

	  if (num_packets > 0) {
	  	struct ip *ip_hdr = (struct ip*)p->buf;
		struct udphdr *udp_hdr = (struct udphdr*)&(p->buf[sizeof(struct ip)]);
		printf("Dest: %s (%u)\n", inet_ntoa(ip_hdr->ip_dst), ntohs(udp_hdr->uh_dport));
		printf("Source: %s (%u)\n", inet_ntoa(ip_hdr->ip_src), ntohs(udp_hdr->uh_sport));
		printf("Next proto: %u\n", ip_hdr->ip_p);
	  }
	}
}


int run_sequential(int argc, char **argv, int sockfd)
{
	printf("Running sequential router code on CPU only\n\n");
	
	unsigned int buf_size = sizeof(packet)*get_batch_size();
	unsigned int results_size = sizeof(int)*get_batch_size();

	// Allocate buffer for packets
	packet* p = (packet *)malloc(buf_size);
	check_malloc(p, "p", __LINE__);

	// Allocate array for results
	int *results = (int*)malloc(results_size);
	check_malloc(results, "results", __LINE__);


	// Run any processing-specific setup code needed
	// (e.g., this might prepare a data structure for LPM)
	setup_sequential();


#ifdef MEASURE_PROCESSING_MICROBENCHMARK
	struct timeval micro_start, micro_stop;
#endif /* MEASURE_PROCESSING_MICROBENCHMARK */

#ifdef MEASURE_BANDWIDTH
	long packets_processed = 0;
	struct timeval bw_start, bw_stop;
	gettimeofday(&bw_start, NULL);
#endif /* MEASURE_BANDWIDTH */

#ifdef MEASURE_LATENCY
	struct timeval lat_start_oldest, lat_start_newest, lat_stop;
	double max_latency, min_latency;
#endif /* MEASURE_LATENCY */


	/* The main loop:
		1) Get a batch of packets
		2) Process them */
	int num_packets;
	while(do_run) {
		
		// Get next batch of packets

#ifdef MEASURE_LATENCY
		gettimeofday(&lat_start_oldest, NULL);
#endif /* MEASURE_LATENCY */
		num_packets = 0;
		while (num_packets == 0) {
			num_packets = get_packets(sockfd, p);
		}
#ifdef MEASURE_LATENCY
		gettimeofday(&lat_start_newest, NULL);
#endif /* MEASURE_LATENCY */
		


		// Process the batch

#ifdef MEASURE_PROCESSING_MICROBENCHMARK
		gettimeofday(&micro_start, NULL);
#endif /* MEASURE_PROCESSING_MICROBENCHMARK */

		printf("Processing %d packets\n\n", num_packets);
		process_packets_sequential(p, results, num_packets);

#ifdef MEASURE_PROCESSING_MICROBENCHMARK
		gettimeofday(&micro_stop, NULL);
		double total_time = (micro_stop.tv_sec - micro_start.tv_sec) * 1000000.0 + (micro_stop.tv_usec - micro_start.tv_usec);

		printf("Performance: %f msec\n", total_time);
#endif /*MEASURE_PROCESSING_MICROBENCHMARK*/

#ifdef MEASURE_LATENCY
		gettimeofday(&lat_stop, NULL);
		max_latency = (lat_stop.tv_sec - lat_start_oldest.tv_sec) * 1000000.0 + (lat_stop.tv_usec - lat_start_oldest.tv_usec);
		min_latency = (lat_stop.tv_sec - lat_start_newest.tv_sec) * 1000000.0 + (lat_stop.tv_usec - lat_start_newest.tv_usec);
		printf("Latencies: Max: %f msec   Min: %f msec\n", max_latency, min_latency);
#endif /* MEASURE_LATENCY */

#ifdef MEASURE_BANDWIDTH
			packets_processed += num_packets;
#endif /* MEASURE_BANDWIDTH */
			
			
		// Print results
		printf("Results:\n");
		int i;
		for (i = 0; i < get_batch_size(); i++) {
			printf("%d, ", results[i]);
		}
		printf("\n\n\n");
	}


#ifdef MEASURE_BANDWIDTH
	// Calculate how many packets we processed per second
	gettimeofday(&bw_stop, NULL);
	double total_time = (bw_stop.tv_sec - bw_start.tv_sec) + (bw_stop.tv_usec - bw_start.tv_usec) / 1000000.0;
	double pkts_per_sec = double(packets_processed) / total_time;	

	printf("Bandwidth: (%ld/%f) = %f packets per second\n", packets_processed, total_time, pkts_per_sec);
#endif /* MEASURE_BANDWIDTH */

	return EXIT_SUCCESS;
}


// Catch Ctrl-C
void sig_handler (int sig)
{
	do_run = false; 
}


/**
 * Program main
 */
int main(int argc, char **argv)
{

	if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
		checkCmdLineFlag(argc, (const char **)argv, "?"))
	{
		printf("Usage -device=n (n >= 0 for deviceID)\n");
		printf("	  -batch=n  (Sets the number of packets in a batch; n > 0)\n");
		printf("	  -wait=n   (Sets how long we wait (milliseconds) for a complete batch of packets; n > 0)\n");
		printf("	  -block=n  (Sets the number of threads in a block; n > 0)\n");
		printf("	  -sequential  (runs router in CPU-only mode w/ sequential code)\n");

		exit(EXIT_SUCCESS);
	}
	
	if (checkCmdLineFlag(argc, (const char **)argv, "batch"))
	{
		int size = getCmdLineArgumentInt(argc, (const char **)argv, "batch");
		set_batch_size(size);
	}
	
	if (checkCmdLineFlag(argc, (const char **)argv, "wait"))
	{
		int wait = getCmdLineArgumentInt(argc, (const char **)argv, "wait");
		set_batch_wait(wait);
	}
	
	int block_size = DEFAULT_BLOCK_SIZE;
	if (checkCmdLineFlag(argc, (const char **)argv, "block"))
	{
		int n = getCmdLineArgumentInt(argc, (const char **)argv, "block");
		if (n > 0) {
			block_size = n;
		}
	}

	// By default, we use device 0, otherwise we override the device ID based on what is provided at the command line
	int devID = 0;

	if (checkCmdLineFlag(argc, (const char **)argv, "device"))
	{
		devID = getCmdLineArgumentInt(argc, (const char **)argv, "device");
		hipSetDevice(devID);
	}

	hipDeviceProp_t deviceProp;
	check_error(hipGetDevice(&devID), "hipGetDevice", __LINE__);
	check_error(hipGetDeviceProperties(&deviceProp, devID), "hipGetDeviceProperties", __LINE__);

	if (deviceProp.computeMode == hipComputeModeProhibited)
	{
		fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
		exit(EXIT_SUCCESS);
	}

	printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
	
	
	// Set up the socket for receiving packets from click
	int sockfd = init_socket();
	if(sockfd == -1) {
	  return -1;
	}

	//test(sockfd);
	
	// Catch Ctrl-C
	signal (SIGQUIT, sig_handler);
	signal (SIGINT, sig_handler);

	// Start the router!
	if (checkCmdLineFlag(argc, (const char **)argv, "sequential"))
	{
		return run_sequential(argc, argv, sockfd);
	}
	else
	{
		return run(argc, argv, block_size, sockfd);
	}
}
