#include <router.h>

#define DEFAULT_BLOCK_SIZE 32


/**
 * Start a loop:
 *	1) Gather packets
 *  2) Copy packets to GPU and process
 *	3) Copy results back and print performance stats
 *
 * We do this with pipelining: while the GPU is processing one buffer of packets,
 * we're copying over the next batch so that it can begin processing them as soon
 * as it finishes processing the first batch.
 */
int run(int argc, char **argv, int block_size, int sockfd)
{
	unsigned int buf_size = sizeof(packet)*get_batch_size();
	unsigned int results_size = sizeof(int)*get_batch_size();

    // Allocate host memory for two batches of up to batch_size packets
	// We will alternate between filling and processing these two buffers
	// (at any given time one of the buffers will either be being filled
	// or being processed)
    packet* h_p1 = (packet *)malloc(buf_size);
	if (h_p1 == NULL) {
		fprintf(stderr, "Failed to allocate packet buffer\n");
		exit(EXIT_FAILURE);
	}
    packet* h_p2 = (packet *)malloc(buf_size);
	if (h_p2 == NULL) {
		fprintf(stderr, "Failed to allocate packet buffer\n");
		exit(EXIT_FAILURE);
	}

	// Allocate host memory for 2 arrays of results
	int *h_results1 = (int*)malloc(results_size);
	if (h_results1 == NULL) {
		fprintf(stderr, "Failed to allocate results array\n");
		exit(EXIT_FAILURE);
	}
	int *h_results2 = (int*)malloc(results_size);
	if (h_results2 == NULL) {
		fprintf(stderr, "Failed to allocate results array\n");
		exit(EXIT_FAILURE);
	}

    // Allocate device memory for up to batch_size packets
	// TODO: wait and allocate only the amount needed after we receive?
    packet *d_p1;
    check_error(hipMalloc((void **) &d_p1, buf_size), "hipMalloc d_p1", __LINE__);
    packet *d_p2;
    check_error(hipMalloc((void **) &d_p2, buf_size), "hipMalloc d_p2", __LINE__);
	// Allocate device memory for results
	int *d_results1;
    check_error(hipMalloc((void **) &d_results1, results_size), "hipMalloc d_results1", __LINE__);
	int *d_results2;
    check_error(hipMalloc((void **) &d_results2, results_size), "hipMalloc d_results2", __LINE__);


	// Receive initial batch of packets to kick off the pipeline
	int num_packets = get_packets(sockfd, h_p1);
	while (num_packets == 0) {
		num_packets = get_packets(sockfd, h_p1);
	}


    // Copy host memory to device
    check_error(hipMemcpy(d_p1, h_p1, buf_size, hipMemcpyHostToDevice), "hipMemcpy (d_p1, h_p1)", __LINE__);


    // Setup execution parameters
	int threads_in_block = block_size;
	int blocks_in_grid = get_batch_size() / threads_in_block;  // FIXME: optimize if we don't have a full batch
	if (get_batch_size() % threads_in_block != 0) {
		blocks_in_grid++;  // need an extra block for the extra threads
	}
   

	// Allocate CUDA events that we'll use for timing
	hipEvent_t start;
	check_error(hipEventCreate(&start), "Create start event", __LINE__);

	hipEvent_t stop;
	check_error(hipEventCreate(&stop), "Create stop event", __LINE__);


    // Perform warmup operation so subsequent CUDA kernel executions have accurate timing
    process_packets_firewall<<< blocks_in_grid, threads_in_block >>>(d_p1, d_results1, num_packets, block_size);
    hipDeviceSynchronize(); // wait for warmup GPU execution to finish



	/* The main loop:
		1) Execute the CUDA kernel
		2) While it's executing, copy the results from the last execution back to the host
		3) While it's executing, copy the packets for the next execution to the GPU
		4) When it finishes executing, print out some timing information   */

	bool first_execution = true;
	packet *h_p_current = h_p1;
	packet *h_p_next = h_p2;
	packet *d_p_current = d_p1;
	packet *d_p_next = d_p2;
	int *h_results_current = h_results1;
	int *h_results_previous = h_results2;
	int *d_results_current = d_results1;
	int *d_results_previous = d_results2;
	
	while(1) {

    	// Record the start event
    	check_error(hipEventRecord(start, NULL), "Record start event", __LINE__);

    	// Execute the kernel
		printf("vvvvv   Begin processing %d packets   vvvvv\n\n", num_packets);
    	process_packets_firewall<<< blocks_in_grid, threads_in_block >>>(d_p_current, d_results_current, num_packets, block_size);

    	// Record the stop event
    	check_error(hipEventRecord(stop, NULL), "Record stop event", __LINE__);


		// TODO: double-check that stuff is really executing when I think it is.
		// I think that calling hipEventRecord(stop) right before this will record
		// when the kernel stops executing, but won't block until this actually happens.
		// The hipEventSynchronize call below does block until the kernel stops.
		// So, I think anything we do here will execute on the CPU while the GPU executes
		// the kernel call we made above.

		if (!first_execution) {
			// Copy the last set of results back from the GPU
    		check_error(hipMemcpy(h_results_previous, d_results_previous, results_size, hipMemcpyDeviceToHost), "hipMemcpy (h_results, d_results)", __LINE__);
		
			// Print results
			printf("Results from last batch:\n");
			int i;
			for (i = 0; i < get_batch_size(); i++) {
				printf("%d, ", h_results_previous[i]);
			}
			printf("\n\n");
		}

		// Get next batch of packets and copy them to the GPU
		// FIXME: We're forcing the results from the current execution to wait
		// until we get the next batch of packets. Is this OK?
		num_packets = get_packets(sockfd, h_p_next);
		while (num_packets == 0) {
			num_packets = get_packets(sockfd, h_p_next);
		}
    	check_error(hipMemcpy(d_p_next, h_p_next, buf_size, hipMemcpyHostToDevice), "hipMemcpy (d_p_next, h_p_next)", __LINE__);




    	// Wait for the stop event to complete (which waits for the kernel to finish)
    	check_error(hipEventSynchronize(stop), "Failed to synchronize stop event", __LINE__);

    	float msecTotal = 0.0f;
    	check_error(hipEventElapsedTime(&msecTotal, start, stop), "Getting time elapsed b/w events", __LINE__);

    	// Compute and print the performance
    	printf(
    	    "Performance= Time= %.3f msec, WorkgroupSize= %u threads/block\n",
    	    msecTotal,
    	    threads_in_block);
		printf("^^^^^   Done processing batch   ^^^^^\n\n\n");




		// Get ready for the next loop iteration
		first_execution = false;
		SWAP(h_p_current, h_p_next, packet*);
		SWAP(d_p_current, d_p_next, packet*);
		SWAP(h_results_current, h_results_previous, int*);
		SWAP(d_results_current, d_results_previous, int*);

	}




    // Clean up memory
    free(h_p1);
	free(h_p2);
    free(h_results1);
	free(h_results2);
    hipFree(d_p1);
	hipFree(d_p2);
    hipFree(d_results1);
	hipFree(d_results2);

    hipDeviceReset();

	return EXIT_SUCCESS;
}

void test(int sockfd) 
{
	printf("Batch Size: %d\n", get_batch_size());
	
	// Initialize a buffer for storing up to batch_size packets
    packet* p = (packet *)malloc(sizeof(packet)*get_batch_size());
    
    while(1) {
      int num_packets = get_packets(sockfd, p);
      printf("i = %d\n", num_packets);

	  if (num_packets > 0) {
	  	struct ip *ip_hdr = (struct ip*)p->buf;
		struct udphdr *udp_hdr = (struct udphdr*)&(p->buf[sizeof(struct ip)]);
		printf("Dest: %s (%u)\n", inet_ntoa(ip_hdr->ip_dst), ntohs(udp_hdr->uh_dport));
		printf("Source: %s (%u)\n", inet_ntoa(ip_hdr->ip_src), ntohs(udp_hdr->uh_sport));
		printf("Next proto: %u\n", ip_hdr->ip_p);
	  }
    }
}


/**
 * Program main
 */
int main(int argc, char **argv)
{
    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
        checkCmdLineFlag(argc, (const char **)argv, "?"))
    {
        printf("Usage -device=n (n >= 0 for deviceID)\n");
        printf("      -batch=n  (Sets the number of packets in a batch; n > 0)\n");
        printf("      -block=n  (Sets the number of threads in a block; n > 0)\n");

        exit(EXIT_SUCCESS);
    }
    
	if (checkCmdLineFlag(argc, (const char **)argv, "batch"))
    {
        int size = getCmdLineArgumentInt(argc, (const char **)argv, "batch");
        set_batch_size(size);
    }
	
	int block_size = DEFAULT_BLOCK_SIZE;
	if (checkCmdLineFlag(argc, (const char **)argv, "block"))
    {
        int n = getCmdLineArgumentInt(argc, (const char **)argv, "block");
		if (n > 0) {
        	block_size = n;
		}
    }

    // By default, we use device 0, otherwise we override the device ID based on what is provided at the command line
    int devID = 0;

    if (checkCmdLineFlag(argc, (const char **)argv, "device"))
    {
        devID = getCmdLineArgumentInt(argc, (const char **)argv, "device");
        hipSetDevice(devID);
    }

    hipDeviceProp_t deviceProp;
    check_error(hipGetDevice(&devID), "hipGetDevice", __LINE__);
    check_error(hipGetDeviceProperties(&deviceProp, devID), "hipGetDeviceProperties", __LINE__);

    if (deviceProp.computeMode == hipComputeModeProhibited)
    {
        fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
        exit(EXIT_SUCCESS);
    }

    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
	
	
	// Set up the socket for receiving packets from click
    int sockfd = init_socket();
    if(sockfd == -1) {
      return -1;
    }

	//test(sockfd);

	// Start the router!
    int result = run(argc, argv, block_size, sockfd);
    exit(result);
}
