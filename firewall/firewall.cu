#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>
#include <netinet/ip.h>
#include <netinet/udp.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>

// Packet collector
#include <packet-collector.h>


#define DEFAULT_BLOCK_SIZE 32

#define RESULT_ERROR -1
#define RESULT_DROP -2
#define RESULT_UNSET -3


/**
 * Checks the supplied cuda error for failure
 */
hipError_t check_error(hipError_t error, char* error_str, int line)
{
	if (error != hipSuccess) {
		fprintf(stderr, "%s returned error (line %d): %s\n", error_str, line, hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
	return error;
}

/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
__global__ void
process_packets(packet *p, int *results, int num_packets, int block_size)
{

	int packet_index = blockIdx.x * block_size + threadIdx.x;
	struct ip *ip_hdr = (struct ip*)p[packet_index].buf;
	if (packet_index < num_packets) {
		results[packet_index] = ip_hdr->ip_p;
	} else {
		results[packet_index] = RESULT_UNSET;
	}


/*
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
         a <= aEnd;
         a += aStep, b += bStep)
    {

        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll

        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            Csub += As[ty][k] * Bs[k][tx];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
	*/
}


/**
 * Start a loop:
 *	1) Gather packets
 *  2) Copy packets to GPU and process
 *	3) Copy results back and print performance stats
 */
int run(int argc, char **argv, int block_size, int sockfd)
{
	unsigned int buf_size = sizeof(packet)*get_batch_size();
	unsigned int results_size = sizeof(int)*get_batch_size();

    // Allocate host memory for up to batch_size packets
    packet* h_p = (packet *)malloc(buf_size);
	if (h_p == NULL) {
		fprintf(stderr, "Failed to allocate packet buffer\n");
		exit(EXIT_FAILURE);
	}
	// Allocate host memory for results
	int *h_results = (int*)malloc(results_size);
	if (h_results == NULL) {
		fprintf(stderr, "Failed to allocate results array\n");
		exit(EXIT_FAILURE);
	}

    // Allocate device memory for up to batch_size packets
	// TODO: wait and allocate only the amount needed after we receive?
    packet *d_p;
    check_error(hipMalloc((void **) &d_p, buf_size), "hipMalloc d_p", __LINE__);
	// Allocate device memory for results
	int *d_results;
    check_error(hipMalloc((void **) &d_results, results_size), "hipMalloc d_results", __LINE__);


	// Receive a batch of packets
	int num_packets = get_packets(sockfd, h_p);
	while (num_packets ==0) {
		num_packets = get_packets(sockfd, h_p);
		printf("Received no packets\n");
	}
	/*if (num_packets >= 0) {
		printf("Received no packets\n");
		return 0;
	}*/


    // Copy host memory to device
    check_error(hipMemcpy(d_p, h_p, buf_size, hipMemcpyHostToDevice), "hipMemcpy (d_p, h_p)", __LINE__);


    // Setup execution parameters
	int threads_in_block = block_size;
	int blocks_in_grid = get_batch_size() / threads_in_block;  // FIXME: optimize if we don't have a full batch
	if (get_batch_size() % threads_in_block != 0) {
		blocks_in_grid++;  // need an extra block for the extra threads
	}


    // Performs warmup operation so subsequent executions have accurate timing
    process_packets<<< blocks_in_grid, threads_in_block >>>(d_p, d_results, num_packets, block_size);
    hipDeviceSynchronize();


    // Allocate CUDA events that we'll use for timing
    hipEvent_t start;
	check_error(hipEventCreate(&start), "Create start event", __LINE__);

    hipEvent_t stop;
    check_error(hipEventCreate(&stop), "Create stop event", __LINE__);

    // Record the start event
    check_error(hipEventRecord(start, NULL), "Record start event", __LINE__);

    // Execute the kernel
    process_packets<<< blocks_in_grid, threads_in_block >>>(d_p, d_results, num_packets, block_size);

    // Record the stop event
    check_error(hipEventRecord(stop, NULL), "Record stop event", __LINE__);

    // Wait for the stop event to complete
    check_error(hipEventSynchronize(stop), "Failed to synchronize stop event", __LINE__);

    float msecTotal = 0.0f;
    check_error(hipEventElapsedTime(&msecTotal, start, stop), "Getting time elapsed b/w events", __LINE__);

    // Compute and print the performance
    float msecPerMatrixMul = msecTotal; //  / nIter;
    printf(
        "Performance= Time= %.3f msec, WorkgroupSize= %u threads/block\n",
        msecPerMatrixMul,
        threads_in_block);

    // Copy result from device to host
    check_error(hipMemcpy(h_results, d_results, results_size, hipMemcpyDeviceToHost), "hipMemcpy (h_results, d_results)", __LINE__);


	// Check for correctness
	bool correct = true;

	int i;
	for (i = 0; i < get_batch_size(); i++) {
		printf("%d, ", h_results[i]);
	}
	printf("\n\n");


    // Clean up memory
    free(h_p);
    free(h_results);
    hipFree(d_p);
    hipFree(d_results);

    hipDeviceReset();


    if (correct)
    {
        return EXIT_SUCCESS;
    }
    else
    {
        return EXIT_FAILURE;
    }
}

void test(int sockfd) 
{
	printf("Batch Size: %d\n", get_batch_size());
	
	// Initialize a buffer for storing up to batch_size packets
    packet* p = (packet *)malloc(sizeof(packet)*get_batch_size());
    
    while(1) {
      int num_packets = get_packets(sockfd, p);
      printf("i = %d\n", num_packets);

	  if (num_packets > 0) {
	  	struct ip *ip_hdr = (struct ip*)p->buf;
		struct udphdr *udp_hdr = (struct udphdr*)&(p->buf[sizeof(struct ip)]);
		printf("Dest: %s (%u)\n", inet_ntoa(ip_hdr->ip_dst), ntohs(udp_hdr->uh_dport));
		printf("Source: %s (%u)\n", inet_ntoa(ip_hdr->ip_src), ntohs(udp_hdr->uh_sport));
		printf("Next proto: %u\n", ip_hdr->ip_p);
	  }
    }
}


/**
 * Program main
 */
int main(int argc, char **argv)
{
    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
        checkCmdLineFlag(argc, (const char **)argv, "?"))
    {
        printf("Usage -device=n (n >= 0 for deviceID)\n");
        printf("      -batch=n  (Sets the number of packets in a batch; n > 0)\n");
        printf("      -block=n  (Sets the number of threads in a block; n > 0)\n");

        exit(EXIT_SUCCESS);
    }
    
	if (checkCmdLineFlag(argc, (const char **)argv, "batch"))
    {
        int size = getCmdLineArgumentInt(argc, (const char **)argv, "batch");
        set_batch_size(size);
    }
	
	int block_size = DEFAULT_BLOCK_SIZE;
	if (checkCmdLineFlag(argc, (const char **)argv, "block"))
    {
        int n = getCmdLineArgumentInt(argc, (const char **)argv, "block");
		if (n > 0) {
        	block_size = n;
		}
    }

    // By default, we use device 0, otherwise we override the device ID based on what is provided at the command line
    int devID = 0;

    if (checkCmdLineFlag(argc, (const char **)argv, "device"))
    {
        devID = getCmdLineArgumentInt(argc, (const char **)argv, "device");
        hipSetDevice(devID);
    }

    hipError_t error;
    hipDeviceProp_t deviceProp;
    error = hipGetDevice(&devID);

    if (error != hipSuccess)
    {
        printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
    }

    error = hipGetDeviceProperties(&deviceProp, devID);

    if (deviceProp.computeMode == hipComputeModeProhibited)
    {
        fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
        exit(EXIT_SUCCESS);
    }

    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
    }
    else
    {
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    }
	
	// Set up the socket for receiving packets from click
    int sockfd = init_socket();
    if(sockfd == -1) {
      return -1;
    }

	//test(sockfd);



	sleep(5);

    int result = run(argc, argv, block_size, sockfd);

    exit(result);
}
